/********************************************************************** 
* DESCRIPTION:
* Serial Concurrent Wave Equation - C Version
* This program implements the concurrent wave equation
 *********************************************************************/
#include <stdio.h> 
#include <stdlib.h>
#include <math.h> 
#include <time.h>
#include <hip/hip_runtime.h>
#define MAXPOINTS 1000000 
#define MAXSTEPS 1000000 
#define MINPOINTS 20 
#define PI 3.14159265

#define thread_num 1008
void check_param(void); 
void init_line(void); 
void update (void); 
void printfinal (void);
int nsteps , tpoints ,rcode;
float values[MAXPOINTS+2],oldval[MAXPOINTS+2], newval[MAXPOINTS+2];
/* number of time steps */
/* total points along string */ /* generic return code */
/* values at time t */
/* values at time (t-dt) */
/* values at time (t+dt) */
/********************************************************************** * Checks input values from parameters *********************************************************************/
void check_param(void) 
{
	char tchar [20];
/* check number of points, number of iterations */
	while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) 
	{
		printf("Enter number of points along vibrating string [%d-%d]: "
			,MINPOINTS , MAXPOINTS); scanf("%s", tchar);
		tpoints = atoi(tchar);
		if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
			printf("Invalid. Please enter value between %d and %d\n", MINPOINTS , MAXPOINTS);
	}
	while ((nsteps < 1) || (nsteps > MAXSTEPS)) 
	{
		printf("Enter number of time steps [1-%d]: ", MAXSTEPS); 
		scanf("%s", tchar);
		

		nsteps = atoi(tchar);
		if ((nsteps < 1) || (nsteps > MAXSTEPS))
			printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
	}
	printf("Using points = %d, steps = %d\n", tpoints, nsteps); 

}

/********************************************************************** * Initialize points on line *********************************************************************/
void init_line(void) 
{
	int  j;
	float x, fac, k, tmp;
/* Calculate initial values based on sine curve */
	fac = 2.0 * PI;
	k = 0.0;
	tmp = tpoints - 1;
	for (j = 1; j <= tpoints; j++) 
	{
		x = k/tmp;
		values[j] = sin (fac * x); 
		k = k + 1.0;

		oldval[j]=values[j];
	}
/* Initialize old values array */
	/*for (i = 1; i <= tpoints; i++) 
		oldval[i] = values[i];*/
}
/********************************************************************** * Calculate new values using wave equation *********************************************************************/
void do_math(int i) 
{
	float dtime, c, dx, tau, sqtau;
	dtime = 0.3;
	c = 1.0;
	dx = 1.0;
	tau = (c * dtime / dx);
	sqtau = tau * tau;
	newval[i] = (2.0 * values[i]) - oldval[i] + (sqtau * (-2.0)*values[i]); 
}

//cuda update
__global__ void cuda_update(int *cuda_nsteps, int *cuda_tpoints,
	float *cuda_values,float *cuda_oldval)
{
	const int idx=blockIdx.x*thread_num+threadIdx.x+1;
	//__shared__ float test_newval[MAXPOINTS+2];
	int i;
	float dtime, c, dx, tau, sqtau;

	const int num=*cuda_tpoints;
	const int step=*cuda_nsteps;
	float reg_values, reg_newval, reg_oldval;
	
	if(idx==1 || idx==num) cuda_values[idx]=0.0;
	else if(idx<*cuda_tpoints)
	{
		reg_oldval=cuda_oldval[idx];
		reg_values=cuda_values[idx];
		
		for(i=1;i<=step;i++)
		{
			dtime = 0.3;
			c = 1.0;
			dx = 1.0;
			tau = (c * dtime / dx);
			sqtau = tau * tau;
			//share_newval[k] = (2.0 * share_values[k]) - share_oldval[k] + (sqtau * (-2.0) * share_values[k]);
			reg_newval=(2.0 * reg_values) - reg_oldval + (sqtau * (-2.0) * reg_values );
			reg_oldval=reg_values;
			reg_values=reg_newval;
		}
		cuda_values[idx]=reg_values;
		
	}






}
/********************************************************************** * Update all values along line a specified number of times *********************************************************************/
void update () 
{
	int i, j;
/* Update values for each time step */
	for (i = 1; i<= nsteps; i++) 
	{

  /* Update points along line for this time step */
		for (j = 1; j <= tpoints; j++) 
		{
/* global endpoints */
			if ((j == 1) || (j == tpoints)) newval[j] = 0.0;
			else do_math(j);
			/* Update old values with new values */
			oldval[j] = values[j]; 
			values[j] = newval[j];
		} 
	}
}
/********************************************************************** * Print final results *********************************************************************/
void printfinal () 
{
	int i;
	for (i = 1; i <= tpoints; i++) 
	{
		printf("%6.4f ", values[i]); if (i%10 == 0)
		printf("\n");
	} 
}
/********************************************************************** * Main program *********************************************************************/
int main(int argc, char *argv[]) 
{
	sscanf(argv[1],"%d",&tpoints); sscanf(argv[2],"%d",&nsteps);
	check_param();
	
	printf("Initializing points on the line...\n"); 
	init_line();
	
	float *cuda_values , *cuda_oldval;
	int *cuda_nsteps, *cuda_tpoints;

	const int block_num=(tpoints+thread_num-1)/thread_num;

	


	/* cuda Malloc */
	hipMalloc((void**) &cuda_values, sizeof(float) * (MAXPOINTS+2));
	hipMalloc((void**) &cuda_oldval, sizeof(float) * (MAXPOINTS+2));
	hipMalloc((void**) &cuda_tpoints, sizeof(int));
	hipMalloc((void**) &cuda_nsteps, sizeof(int));

	/* cuda Memory copy */
	hipMemcpy(cuda_values, values, sizeof(float) * (MAXPOINTS+2), hipMemcpyHostToDevice);
	hipMemcpy(cuda_oldval, oldval, sizeof(float) * (MAXPOINTS+2), hipMemcpyHostToDevice);
	hipMemcpy(cuda_nsteps, &nsteps, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cuda_tpoints, &tpoints, sizeof(int), hipMemcpyHostToDevice);

	//cuda_init_line<<<block_num,thread_num>>>(cuda_tpoints,cuda_values,cuda_oldval);

	//cudaMemcpy(values,cuda_values,sizeof(float) * (MAXPOINTS+2), cudaMemcpyDeviceToHost);
	
	//printfinal();
	
	printf("Updating all points for all time steps...\n");
	
	cuda_update<<<block_num,thread_num>>>(cuda_nsteps,cuda_tpoints,cuda_values,cuda_oldval);


	//return result
	hipMemcpy(values, cuda_values, sizeof(float) * (MAXPOINTS+2), hipMemcpyDeviceToHost);

	//update();
	
	printf("Printing final results...\n");
	printfinal();
	printf("\nDone.\n\n"); return 0;
}